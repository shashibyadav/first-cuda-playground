
#include <hip/hip_runtime.h>
#include <iostream>

// Kernel definition

__global__ void VecAdd(float* A, float* B, float* C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    std::cout << "Hello, World!" << std::endl;
    int N = 10;
    float* A = new float[N];
    float* B = new float[N];
    float* C = new float[N];

    for (int i = 0; i < N; ++i) {
        A[i] = 1;
        B[i] = 1;
    }

    // Allocate memory on the device (GPU)
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    VecAdd<<<(N + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock>>>(d_A, d_B, d_C);

    hipDeviceSynchronize();

    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        std::cout << "Result " << i << " :- " << C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
